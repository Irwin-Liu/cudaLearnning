
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hip/hip_runtime_api.h>

__global__ void HelloWorldDynamicKernel(const int block_size, int depth)
{
    int tid = threadIdx.x;
    printf("Recursion=%d: Hello World from thread %d block %d\n", depth, tid,
           blockIdx.x);

    // condition to stop recursive execution
    if (block_size == 1)
                return;

    // reduce block size to half
    int next_block_size = block_size >> 1;

    // thread 0 launches child grid recursively
    if((tid == 0) && (next_block_size > 0))
    {
        HelloWorldDynamicKernel<<<1, next_block_size>>>(next_block_size, ++depth);
        //cudaDeviceSynchronize();
        printf("-------> execution depth: %d\n", depth);
    }
}

int main(int argc, char **argv) {
    const int block_size = 8;
    const int grid_size  = 1;

    std::printf("Excuting GPU:\n");

    HelloWorldDynamicKernel<<<grid_size, block_size>>>(block_size, 0);

    hipDeviceSynchronize();

    return  0;
}

