
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include "hip/hip_runtime_api.h"

double *InitializeArray(const int length, const int seed) {
    double *A = (double*)malloc(length * sizeof(double));

    std::default_random_engine e;
    std::uniform_real_distribution<double> rand(0, 10);
    e.seed(seed);

    for (int i = 0; i < length; ++i) {
        A[i] = rand(e);
    }

    return A;
}

void PrintArray(double *A, const int length, std::string str) {
    std::cout <<"Array " << str << ":";

    for (int i = 0; i < length; ++i) {
        std::cout << " " << A[i];
    }

    std::cout << std::endl;
}

__host__ __device__ double MaxElement(double a, double b, double c) {
    if ((a >= b) && (a >= c)) {
        return a;
    }
    if (b >= c) {
        return b;
    }
    return c;
}

double *MaxElements(double *A, double *B, double *C, const int length) {
    double *D = (double*)malloc(length * sizeof(double));

    for (int i = 0; i < length; ++i) {
        D[i] = MaxElement(A[i], B[i], C[i]);
    }

    return D;
}

__global__ void MaxElementsKernel(double *A, double *B, double *C, double *D) {
    int i = threadIdx.x;

    D[i] = MaxElement(A[i], B[i], C[i]);
}

int main() {
    const int length = 10;
    const size_t size = length * sizeof(double);

    double *h_A, *h_B, *h_C, *h_D;
    h_A = (double*)malloc(size);
    h_B = (double*)malloc(size);
    h_C = (double*)malloc(size);
    h_D = (double*)malloc(size);

    h_A = InitializeArray(length, 0);
    h_B = InitializeArray(length, 5);
    h_C = InitializeArray(length, 10);

    PrintArray(h_A, length, "A");
    PrintArray(h_B, length, "B");
    PrintArray(h_C, length, "C");

    std::cout << "CPU Result:\n";
    PrintArray(MaxElements(h_A, h_B, h_C, length), length, "D");

    std::cout << "GPU Result:\n";
    const int dev = 0;
    hipSetDevice(dev);

    double *d_A, *d_B, *d_C, *d_D;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    hipMalloc((void **)&d_D, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

    MaxElementsKernel<<<1, length>>>(d_A, d_B, d_C, d_D);

    hipMemcpy(h_D, d_D, size, hipMemcpyDeviceToHost);

    PrintArray(h_D, length, "D");

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);

    return 0;
}

