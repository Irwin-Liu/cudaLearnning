
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void checkIndexKernel() {
    int threadID = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int blockID = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    int threadIDinGird = threadID + blockID * blockDim.x * blockDim.y * blockDim.z;

    printf("thread id in grid: %2d; thread id in block: %2d (%d, %d, %d) in blockDim (%d, %d, %d); block id: %d (%d, %d, %d) in gridDim (%d, %d, %d)\n",
           threadIDinGird,
           threadID, threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z,
           blockID, blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
}

int main() {
    dim3 block_size(3, 2, 2);
    dim3 grid_size(2);

    checkIndexKernel<<<grid_size, block_size>>>();
    hipDeviceSynchronize();

    return 0;
}