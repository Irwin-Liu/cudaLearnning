
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void helloWorldKernel() {
    printf("Hello World from GPU\n");
}

__global__ void helloWorldwithThreadKernel() {
    printf("Hello World from GPU block: %d thread: %d\n", blockIdx.x, threadIdx.x);
}

int main() {
    std::printf("Hello World from CPU\n");
    std::printf("--------------------------------\n");

    helloWorldKernel<<<1, 10>>>();
    hipDeviceSynchronize();
    std::printf("--------------------------------\n");

    helloWorldwithThreadKernel<<<4, 2>>>();
    hipDeviceSynchronize();
    std::printf("--------------------------------\n");


    return 0;
}
