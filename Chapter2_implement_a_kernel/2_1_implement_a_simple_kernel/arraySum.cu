
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include "hip/hip_runtime_api.h"

double *InitializeArray(const int length, const int seed) {
    double *A = (double*)malloc(length * sizeof(double));

    std::default_random_engine e;
    std::uniform_real_distribution<double> rand(0, 10);
    e.seed(seed);

    for (int i = 0; i < length; ++i) {
        A[i] = rand(e);
    }

    return A;
}

void PrintArray(double *A, const int length, std::string str) {
    std::cout <<"Array " << str << ":";

    for (int i = 0; i < length; ++i) {
        std::cout << " " << A[i];
    }

    std::cout << std::endl;
}

double *ArraySum(double *A, double *B, const int length) {
    double *C = (double*)malloc(length * sizeof(double));

    for (int i = 0; i < length; ++i) {
        C[i] = A[i] + B[i];
    }

    return C;
}

__global__ void ArraySumKernel(double *A, double *B, double *C) {
    int i = threadIdx.x;

    C[i] = A[i] + B[i];
}

int main() {
    const int length = 10;
    const size_t size = length * sizeof(double);

    double *h_A, *h_B, *h_C;
    h_A = (double*)malloc(size);
    h_B = (double*)malloc(size);
    h_C = (double*)malloc(size);

    h_A = InitializeArray(length, 0);
    h_B = InitializeArray(length, 10);

    PrintArray(h_A, length, "A");
    PrintArray(h_B, length, "B");

    std::cout << "CPU Array Sum Result:\n";
    PrintArray(ArraySum(h_A, h_B, length), length, "C");

    std::cout << "GPU Array Sum Result:\n";
    const int dev = 0;
    hipSetDevice(dev);

    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    ArraySumKernel<<<1, length>>>(d_A, d_B, d_C);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    PrintArray(h_C, length, "C");

    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}