
#include <hip/hip_runtime.h>
__device__ int count = 0;
 
__global__ static void sum(int* data_gpu, int* block_gpu, int *sum_gpu, int length)
{
  extern __shared__ int blocksum[];
  __shared__ int islast;
  int offset;
 
  const int tid  = threadIdx.x;
  const int bid  = blockIdx.x;
  const int tnum = blockDim.x;
  const int bnum = gridDim.x;
  blocksum[tid]  = 0;

  for (int i = bid * tnum + tid; i < length; i += bnum * tnum) {
    blocksum[tid] += data_gpu[i];
  } 
 
  __syncthreads();

  offset = tnum / 2;
  while (offset > 0) {
    if(tid < offset) {
      blocksum[tid] += blocksum[tid + offset];
    }
    offset >>= 1;
    __syncthreads();
  }
 
  if (tid == 0) {
    block_gpu[bid] = blocksum[0];
    __threadfence();
 
    int value = atomicAdd(&count, 1);
    islast = (value == gridDim.x - 1);
  }
 
  __syncthreads();
 
  if (islast) {
    if (tid == 0) {
      int s = 0;
      for (int i = 0; i < bnum; i++) {
        s += block_gpu[i];
      }
      *sum_gpu = s;
    }
  }
}
