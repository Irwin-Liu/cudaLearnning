#include "hip/hip_runtime.h"
__global__ void myKernel1(int* result)
{
    int tid = threadIdx.x;
    int ref1 = myArray[tid] * 1;

    myArray[tid + 1] = 2;

    int ref2 = myArray[tid] * 1;
    result[tid] = ref1 * ref2;
}

__global__ void myKernel2(int* result)
{
    __shared__ volatile float myArray[512];

    int tid = threadIdx.x;
    int ref1 = myArray[tid] * 1;

    myArray[tid + 1] = 2;

    __syncthreads();

    int ref2 = myArray[tid] * 1;
    result[tid] = ref1 * ref2;
}
