
#include <hip/hip_runtime.h>
__global__ void transpose(float **inputdata, float **outputdata, int width, int height)
{
  __shared__ float block[32][32]; // alloc static shared memory, blockDim.x = 32, blockDim.y = 32

  // read matrix to shared memory
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((x >= width) || (y >= height)) {
    return;
  }

  int i = threadIdx.x, j = threadIdx.y;

  block[j][i] = inputdata[x][y];

  __syncthreads();

  outputdata[x][y] = block[i][j];
}
