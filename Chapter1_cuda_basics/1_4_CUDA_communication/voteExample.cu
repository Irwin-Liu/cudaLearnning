
#include <hip/hip_runtime.h>
__global__ void vote_all(int* a, int* b, int n) 
{ 
    int tid = threadIdx.x; 
    if (tid > n) { 
       return; 
    } 
    int temp = a[tid]; 
    b[tid] = __all(temp > 100); 
} 
 
__global__ void vote_any(int* a, int* b, int n) 
{ 
    int tid = threadIdx.x; 
    if (tid > n) { 
       return; 
    } 
    int temp = a[tid]; 
    b[tid] = __any(temp > 100); 
}
