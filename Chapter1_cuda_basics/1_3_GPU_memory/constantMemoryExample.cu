#include "hip/hip_runtime.h"
__constant__ int  num = 16; // method 1
__constant__ int  con_t[16] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15}; // method 1
__constant__ char con_p[16]; // method 2

__global__ static void constantMemoryDemo(char* result)
{
  int index = threadIdx.x;
  if (index >= num) {
    return;
  }
  result[index] = con_p[index] + con_t[index];
}

int main(int argc, char** argv[])
{
  char h_result[16] = {0};
  char *d_result;
  hipMalloc((void **)&d_result, 16 * sizeof(char));

  char h_con_p[16] = "Let's learn cuda";
  hipMemcpyToSymbol(HIP_SYMBOL(con_p), h_con_p, 16 * sizeof(char)); // method 2
  constantMemoryDemo<<<1, 32>>>(d_result);
  hipMemcpy(&h_result, d_result, 16 * sizeof(char), hipMemcpyDeviceToHost);

  hipFree(d_result);
}
