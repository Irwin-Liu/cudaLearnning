#include "hip/hip_runtime.h"
__constant__ char con_p[16];
__constant__ int  con_t[16] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15};
__constant__ int  num = 16;

__global__ static void constantMemoryDemo(char* result)
{
  int index = threadIdx.x;
  if (index >= num) {
    return;
  }
  result[index] = con_p[index] + con_t[index];
}

int main(int argc, char** argv[])
{
  char h_result[16] = {0};
  char *d_result;
  hipMalloc((void **)&d_result, 16 * sizeof(char));

  char h_con_p[16] = "Let's learn cuda";
  hipMemcpyToSymbol(HIP_SYMBOL(con_p), h_con_p, 16 * sizeof(char));
  constantMemoryDemo<<<1, 32>>>(d_result);
  hipMemcpy(&h_result, d_result, 16 * sizeof(char), hipMemcpyDeviceToHost);

  hipFree(d_result);
}
