#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <iostream>
#include <ctime>
#include <stdio.h>

using namespace std;

// step 1: alloc texture memory
texture<float, 1, hipReadModeElementType> tex1D_load;

__global__ void kernel(float *d_out, int size)
{
    int index;
    index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        d_out[index] = tex1Dfetch(tex1D_load, index); // step 3: get values of texture memory
        printf("%f\n", d_out[index]);
    }
}

int main(int argc, char** argv[])
{
    int size = 120;
    size_t Size = size * sizeof(float);
    float *harray;
    float *d_in;
    float *d_out;

    harray = new float[size];
    hipMalloc((void **)&d_out, Size);
    hipMalloc((void **)&d_in, Size);

    //initial host memory
    for (int i = 0; i < 4; m++)
    {
        printf("i = %d\n", i);

        for (int loop = 0; loop < size; loop++)
        {
            harray[loop] = loop + i * 1000;
        }

        //copy to d_in
        hipMemcpy(d_in, harray, Size, hipMemcpyHostToDevice);

        //step 2: bind texture
        hipBindTexture(0, tex1D_load, d_in, Size); // 0 means no offset

        int nBlocks = (Size - 1) / 128 + 1;
        kernel<<<nBlocks, 128>>>(d_out, size); // step 3

        // step 4 unbind texture
        hipUnbindTexture(tex1D_load);

        hipDeviceSynchronize();
    }

    delete[] harray;
    hipUnbindTexture(&tex1D_load);
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
