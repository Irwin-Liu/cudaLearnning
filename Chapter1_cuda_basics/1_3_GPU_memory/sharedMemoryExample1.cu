#include "hip/hip_runtime.h"
// dynamic alloc
__global__ void sharedMemoryDemo1( )
{
  extern __shared__ float shared_data_dynamic[];
  float *data = (float*)shared_data_dynamic;
 
  int id = threadIDx.x;
  data[id] = 0.0f; // initialization
}
 
// static alloc
__global__ void sharedMemoryDemo2( )
{
  __shared__ int shared_data_static[16];
 
  int id = threadIDx.x;
  shared_data_static[id] = 0; // initialization
}
 
int main(int argc, char** argv)
{
  int length = 16;
  sharedMemoryDemo1<<<1, length, length * sizeof(float)>>>();
  sharedMemoryDemo2<<<1, length>>>();
 
  return 0;
}
