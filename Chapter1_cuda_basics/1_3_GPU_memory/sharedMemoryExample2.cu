#include "hip/hip_runtime.h"
__global__ void sharedMemoryDemo3( )
{
  extern __shared__ char shared_data[];
 
  double* data1 = (double*)shared_data;
  float*  data2 = (float*)&data1[128];
  int*    data3 = (int*)&data2[64];
 
  // initialization
  int id = threadIDx.x;
  if (id < 128) {
    data1[id] = 0.0f;
  }
  if (id < 64) {
    data2[id] = 0.0f;
  }
  data3[id] = 0;
}
 
int main(int argc, char** argv)
{
  // alloc these arrays on GPU shared memory
  double data1[128];
  float  data2[64];
  int    data3[256];
 
  sharedMemoryDemo3<<<1, 256, 128 * sizeof(double) + 64 * sizeof(float) + 256 * sizeof(int)>>>();
}
