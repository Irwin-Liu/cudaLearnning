#include "hip/hip_runtime.h"
__global__ void globalMemoryDemo1(float* A)
{
  int i = threadIDx.x;
  A[i] *= 2.0f;
}
 
__global__ void globalMemoryDemo2(float* B, int pitch, int width, int height)
{
  for (int r = 0; r < height; r++) {
    float* row = (float*)((char*)B + r * pitch);
    for (int c = 0; c < width; c++) {
      float tmp = row[c];
    }
  }
}
 
__global__ void globalMemoryDemo3(hipPitchedPtr pitched_D, hipExtent extent)
{
  char* D = pitched_D.ptr;
  size_t pitch = pitched_D.pitch;
  size_t slice_pitch = pitch * extent.height;
  for (int k = 0; k < extent.depth; k++) {
    char* slice = D + k * slice_pitch;
    for (int j = 0; j < extent.height; j++) {
      float* row = (float*)(slice + y * pitch);
      for (int i = 0; i < extent.width; i++) {
        float tmp = row[i];
      }
    }
  }
}
 
int main(int argc, char** argv)
{
  // 1D array
  const int N = 256;
  size_t size = N * sizeof(float);
  float *h_A, *d_A;
 
  h_A = (float *)malloc(size); // alloc on host memory
  for (int i = 0; i < N; i++) {
    h_A[i] = i;
  }
 
  hipMalloc((void **)&d_A, size); // alloc on GPU global memory
 
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice); // copy data from host to device
  globalMemoryDemo1<<<1, N>>>(d_A); // start kernel, change elements in d_A
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost); // copy data from device back to host
 
  hipFree(d_A); //release global memory
  free(h_A);
 
  // 2D array
  int pitch, width = 32, height = 16;
  float *d_B, *d_C;
 
  hipMallocPitch((void **)&d_B, &pitch, width * sizeof(float), height); // alloc on GPU global memory
  globalMemoryDemo2<<<1, 1>>>(d_B, pitch, width, height); // start kernel, traverse elements of d_B
  hipMemcpy2DToArray(d_C, 0, 0, d_B, pitch, width * sizeof(float), height, hipMemcpyDeviceToDevice); //copy data from d_B to d_C
 
  hipFree(d_B);
  hipFree(d_C);
 
  // 3D array
  hipPitchedPtr d_D;
  hipExtent extent = make_hipExtent(64, 32, 16); // define dimension variable extent
 
  hipMalloc3D(&d_D, extent); // alloc on GPU global memory
  globalMemoryDemo3<<<1, 1>>>(d_D, extent); // start kernel, traverse elements of d_D
 
  hipFree(d_D);
}
