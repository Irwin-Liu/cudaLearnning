int main(int argc, char** argv[])
{
  const int n = 64;
  size_t size = n * sizeof(int);

  int *data1, *data2, *data3, *data4;
  int *d_data4;

  // alloc pinned memory
  hipHostAlloc((void **)&data1, size, hipHostMallocDefault);

  // alloc portable memory
  hipHostAlloc((void **)&data2, size, hipHostMallocPortable);

  // alloc write-combined memory
  hipHostAlloc((void **)&data3, size, hipHostMallocWriteCombined);

  // alloc mapped memory
  hipDeviceProp_t prop;
  int flag, device_id = 0;
  hipGetDeviceProperties(prop, device_id);
  if (prop.canMapHostMemory) {
    hipHostAlloc((void **)&data4, size, hipHostMallocMapped);
    hipHostGetDevicePointer(&d_data4, data4, flag);
  }
  
  // free
  hipHostFree(data1);
  hipHostFree(data2);
  hipHostFree(data3);
  if (prop.canMapHostMemory) {
    hipHostFree(data4);
    hipFree(d_data4);
  }

  return 0;
}
