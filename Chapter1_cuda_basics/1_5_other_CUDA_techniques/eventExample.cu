int main(int argc, char** argv)
{
  // create two events
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // record start event on the default stream
  hipEventRecord(start);

  // execute kernel
  testkernel<<<1, 1024>>>();

  // record stop event on the default stream
  hipEventRecord(stop);

  // wait until the stop event completes
  hipEventSynchronize(stop);

  // calculate the elapsed time between two events
  float time;
  hipEventElapsedTime(&time, start, stop);

  // clean up the two events
  hipEventDestroy(start);
  hipEventDestroy(stop);
}
