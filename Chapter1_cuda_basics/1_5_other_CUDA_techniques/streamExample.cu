int main(int argc, char** argv)
{
  // create 2 streams
  hipStream_t stream[2];
  // initialize cuda stream
  for (int i = 0; i < 2; i++) {
    hipStreamCreate(&stream[i]);
  }

  // alloc pinned memory and device memory;
  float *hostPtr, *d_in, *d_out;
  size_t size = 512 * sizeof(float);
  hipHostMalloc((void **)hostPtr, 2 * size);
  hipMalloc((void **)d_in,  2 * size);
  hipMalloc((void **)d_out, 2 * size);

  // copy data from host to device in 2 streams
  for (int i = 0; i < 2; i++) {
    hipMemcpyAsync(d_in + i * size, hostPtr + i * size, size, cudaMemcpyHostToDevce, stream[i]);
  }

  // run testkernel in 2 streams
  for (int i = 0; i < 2; i++) {
    testkernel<<<1, 512, 0, stream[i]>>>(d_in + i * size, d_out + i * size, size);
  }

  // copy data from device to host in 2 streams
  for (int i = 0; i < 2; i++) {
    hipMemcpyAsync(hostPtr + i * size, d_out + i * ptr, size, cudaMemcpyDevceToHost, stream[i]);
  }

  hipDeviceSynchronize();

  // release cuda stream
  for (int i = 0; i < 2; i++) {
    hipStreamDestroy(stream[i]);
  }
  hipFree(hostPtr);

  return 0;
}
