#include "hip/hip_runtime.h"
// conditional assignment
if (a == b) {
  c++;
}
// instead “==” by “-”
c += !(a - b)
 
 
// segmentation judgment
if (a > b) {
  a = b;
}
// use value of (a > b)
a = a - (a > b) * (a - b);
 
 
// min(a, b)
if (a < b) {
  c = a;
} else {
  c = b;
}
// reduce time cost of min(a, b)
c = (a < b) * a + (a >= b) * b;
 
 
// assignment according to control condition
// input a = 0 or 1
if (a == 0) {
  b = 1;
} else {
  b = 5;
}
// rewrite according a = 0 or 1
b = (a << 2) + 1;
