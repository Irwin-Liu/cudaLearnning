
#include <hip/hip_runtime.h>
struct S1_t { static const int value = 4; };
template <int X, typename T2>

__device__ void foo(int *p1, int *p2) { 
// no argument specified, loop will be completely unrolled
#pragma unroll
for (int i = 0; i < 12; ++i)
  p1[i] += p2[i] * 2;
   
// unroll value = 8
#pragma unroll (X+1)
for (int i = 0; i < 12; ++i)
  p1[i] += p2[i] * 4;
 
// unroll value = 1, loop unrolling disabled
#pragma unroll 1
for (int i = 0; i < 12; ++i)
  p1[i] += p2[i] * 8;
 
// unroll value = 4
#pragma unroll (T2::value)
for (int i = 0; i < 12; ++i)
  p1[i] += p2[i] * 16;
}
 
__global__ void bar(int *p1, int *p2) {
  foo<7, S1_t>(p1, p2);
}
